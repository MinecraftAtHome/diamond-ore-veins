#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <inttypes.h>
#include <math.h>



#define __STDC_FORMAT_MACROS 1

#include <stdlib.h>
#include <stddef.h>
#include <inttypes.h>

#ifdef BOINC
  #include "boinc_api.h"
#if defined _WIN32 || defined _WIN64
  #include "boinc_win.h"
#endif
#endif

#define GPU_ASSERT(code) gpuAssert((code), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s (code %d) %s %d\n", hipGetErrorString(code), code, file, line);
    exit(code);
  }
}

///=============================================================================
///                      Compiler and Platform Features
///=============================================================================

typedef int8_t      i8;
typedef uint8_t     u8;
typedef int16_t     i16;
typedef uint16_t    u16;
typedef int32_t     i32;
typedef uint32_t    u32;
typedef int64_t     i64;
typedef uint64_t    u64;
typedef float       f32;
typedef double      f64;


#define STRUCT(S) typedef struct S S; struct S

#if __GNUC__

#define IABS(X)                 __builtin_abs(X)
#define PREFETCH(PTR,RW,LOC)    __builtin_prefetch(PTR,RW,LOC)
#define likely(COND)            (__builtin_expect(!!(COND),1))
#define unlikely(COND)          (__builtin_expect((COND),0))
#define ATTR(...)               __attribute__((__VA_ARGS__))
#define BSWAP32(X)              __builtin_bswap32(X)
#define UNREACHABLE()           __builtin_unreachable()

#else

#define IABS(X)                 ((int)abs(X))
#define PREFETCH(PTR,RW,LOC)
#define likely(COND)            (COND)
#define unlikely(COND)          (COND)
#define ATTR(...)
__device__ __host__ static inline uint32_t BSWAP32(uint32_t x) {
    x = ((x & 0x000000ff) << 24) | ((x & 0x0000ff00) <<  8) |
        ((x & 0x00ff0000) >>  8) | ((x & 0xff000000) >> 24);
    return x;
}
#if _MSC_VER
#define UNREACHABLE()           
#else
#define UNREACHABLE()           exit(1) // [[noreturn]]
#endif

#endif

/// imitate amd64/x64 rotate instructions

__device__ __host__ static inline ATTR(const, always_inline, artificial)
uint64_t rotl64(uint64_t x, uint8_t b)
{
    return (x << b) | (x >> (64-b));
}

__device__ __host__ static inline ATTR(const, always_inline, artificial)
uint32_t rotr32(uint32_t a, uint8_t b)
{
    return (a >> b) | (a << (32-b));
}

/// integer floor divide
__device__ __host__ static inline ATTR(const, always_inline)
int32_t floordiv(int32_t a, int32_t b)
{
    int32_t q = a / b;
    int32_t r = a % b;
    return q - ((a ^ b) < 0 && !!r);
}

///=============================================================================
///                    C implementation of Java Random
///=============================================================================

__device__ __host__ static inline void setSeed(uint64_t *seed, uint64_t value)
{
    *seed = (value ^ 0x5deece66d) & ((1ULL << 48) - 1);
}

__device__ __host__ static inline int next(uint64_t *seed, const int bits)
{
    *seed = (*seed * 0x5deece66d + 0xb) & ((1ULL << 48) - 1);
    return (int) ((int64_t)*seed >> (48 - bits));
}

__device__ __host__ static inline int nextInt(uint64_t *seed, const int n)
{
    int bits, val;
    const int m = n - 1;

    if ((m & n) == 0) {
        uint64_t x = n * (uint64_t)next(seed, 31);
        return (int) ((int64_t) x >> 31);
    }

    do {
        bits = next(seed, 31);
        val = bits % n;
    }
    while (bits - val + m < 0);
    return val;
}

__device__ __host__ static inline uint64_t nextLong(uint64_t *seed)
{
    return ((uint64_t) next(seed, 32) << 32) + next(seed, 32);
}

__device__ __host__ static inline float nextFloat(uint64_t *seed)
{
    return next(seed, 24) / (float) (1 << 24);
}

__device__ __host__ static inline double nextDouble(uint64_t *seed)
{
    uint64_t x = (uint64_t)next(seed, 26);
    x <<= 27;
    x += next(seed, 27);
    return (int64_t) x / (double) (1ULL << 53);
}

/* A macro to generate the ideal assembly for X = nextInt(*S, 24)
 * This is a macro and not an inline function, as many compilers can make use
 * of the additional optimisation passes for the surrounding code.
 */
#define JAVA_NEXT_INT24(S,X)                \
    do {                                    \
        uint64_t a = (1ULL << 48) - 1;      \
        uint64_t c = 0x5deece66dULL * (S);  \
        c += 11; a &= c;                    \
        (S) = a;                            \
        a = (uint64_t) ((int64_t)a >> 17);  \
        c = 0xaaaaaaab * a;                 \
        c = (uint64_t) ((int64_t)c >> 36);  \
        (X) = (int)a - (int)(c << 3) * 3;   \
    } while (0)


/* Jumps forwards in the random number sequence by simulating 'n' calls to next.
 */
__device__ __host__ static inline void skipNextN(uint64_t *seed, uint64_t n)
{
    uint64_t m = 1;
    uint64_t a = 0;
    uint64_t im = 0x5deece66dULL;
    uint64_t ia = 0xb;
    uint64_t k;

    for (k = n; k; k >>= 1)
    {
        if (k & 1)
        {
            m *= im;
            a = im * a + ia;
        }
        ia = (im + 1) * ia;
        im *= im;
    }

    *seed = *seed * m + a;
    *seed &= 0xffffffffffffULL;
}


///=============================================================================
///                               Xoroshiro 128
///=============================================================================

STRUCT(Xoroshiro)
{
    uint64_t lo, hi;
};

__device__ __host__ static inline void xSetSeed(Xoroshiro *xr, uint64_t value)
{
    const uint64_t XL = 0x9e3779b97f4a7c15ULL;
    const uint64_t XH = 0x6a09e667f3bcc909ULL;
    const uint64_t A = 0xbf58476d1ce4e5b9ULL;
    const uint64_t B = 0x94d049bb133111ebULL;
    uint64_t l = value ^ XH;
    uint64_t h = l + XL;
    l = (l ^ (l >> 30)) * A;
    h = (h ^ (h >> 30)) * A;
    l = (l ^ (l >> 27)) * B;
    h = (h ^ (h >> 27)) * B;
    l = l ^ (l >> 31);
    h = h ^ (h >> 31);
    xr->lo = l;
    xr->hi = h;
}

__device__ __host__ static inline uint64_t xNextLong(Xoroshiro *xr)
{
    uint64_t l = xr->lo;
    uint64_t h = xr->hi;
    uint64_t n = rotl64(l + h, 17) + l;
    h ^= l;
    xr->lo = rotl64(l, 49) ^ h ^ (h << 21);
    xr->hi = rotl64(h, 28);
    return n;
}

__device__ __host__ static inline int xNextInt(Xoroshiro *xr, uint32_t n)
{
    uint64_t r = (xNextLong(xr) & 0xFFFFFFFF) * n;
    if ((uint32_t)r < n)
    {
        while ((uint32_t)r < (~n + 1) % n)
        {
            r = (xNextLong(xr) & 0xFFFFFFFF) * n;
        }
    }
    return r >> 32;
}

__device__ __host__ static inline double xNextDouble(Xoroshiro *xr)
{
    return (xNextLong(xr) >> (64-53)) * 1.1102230246251565E-16;
}

__device__ __host__ static inline float xNextFloat(Xoroshiro *xr)
{
    return (xNextLong(xr) >> (64-24)) * 5.9604645E-8F;
}

__device__ __host__ static inline void xSkipN(Xoroshiro *xr, int count)
{
    while (count --> 0)
        xNextLong(xr);
}

__device__ __host__ static inline uint64_t xNextLongJ(Xoroshiro *xr)
{
    int32_t a = xNextLong(xr) >> 32;
    int32_t b = xNextLong(xr) >> 32;
    return ((uint64_t)a << 32) + b;
}

__device__ __host__ static inline int xNextIntJ(Xoroshiro *xr, uint32_t n)
{
    int bits, val;
    const int m = n - 1;

    if ((m & n) == 0) {
        uint64_t x = n * (xNextLong(xr) >> 33);
        return (int) ((int64_t) x >> 31);
    }

    do {
        bits = (xNextLong(xr) >> 33);
        val = bits % n;
    }
    while (bits - val + m < 0);
    return val;
}

typedef struct {
    Xoroshiro internal;
    int num_calls;
} RNG; 

#define XRSR_MIX1          0xbf58476d1ce4e5b9
#define XRSR_MIX2          0x94d049bb133111eb
#define XRSR_MIX1_INVERSE  0x96de1b173f119089
#define XRSR_MIX2_INVERSE  0x319642b2d24d8ec3
#define XRSR_SILVER_RATIO  0x6a09e667f3bcc909
#define XRSR_GOLDEN_RATIO  0x9e3779b97f4a7c15

__device__ __host__   uint64_t mix64(uint64_t a) {
	a = (a ^ a >> 30) * XRSR_MIX1;
	a = (a ^ a >> 27) * XRSR_MIX2;
	return a ^ a >> 31;
}

__device__ __host__  RNG rng_new() {
    return (RNG){.internal=(Xoroshiro){0}};
}

__device__ __host__  static void rng_set_seed(RNG *rng, uint64_t seed) {
    seed ^= XRSR_SILVER_RATIO;
    rng->internal.lo = mix64(seed);
    rng->internal.hi = mix64(seed + XRSR_GOLDEN_RATIO);
}

__device__ __host__  static void rng_set_internal(RNG *rng, uint64_t lo, uint64_t hi) {
    rng->internal.lo = lo;
    rng->internal.hi = hi;
}

__device__ __host__  static uint64_t rng_next(RNG *rng, int32_t bits) {
    rng->num_calls++;
    return xNextLong(&rng->internal) >> (64 - bits);
}

__device__ __host__  static int32_t rng_next_int(RNG *rng, uint32_t bound) {
    uint32_t r = rng_next(rng, 31);
    uint32_t m = bound - 1;
    if ((bound & m) == 0) {
        // (int)((long)p_188504_ * (long)this.next(31) >> 31);
        r = (uint32_t)((uint64_t)bound * (uint64_t)r >> 31);
    }
    else {
        for (uint32_t u = r; (int32_t)(u - (r = u % bound) + m) < 0; u = rng_next(rng, 31));
    }
    return r;
}

__device__ __host__  static float rng_next_float(RNG *rng) {
    return xNextFloat(&rng->internal);
}

__device__ __host__  static double rng_next_double(RNG *rng) { // whoops!
    int32_t i = rng_next(rng, 26);
    int32_t j = rng_next(rng, 27);
    uint64_t k = ((uint64_t)i << 27) + (uint64_t)j;
    return (double)k * (double)1.110223E-16F;
}

__device__ __host__  static int rng_next_between_inclusive(RNG *rng, int i, int j) {
    return rng_next_int(rng, j - i + 1) + i;
}

__device__ __host__  static uint64_t rng_next_long(RNG *rng) {
    int32_t i = rng_next(rng, 32);
    int32_t j = rng_next(rng, 32);
    uint64_t k = (uint64_t)i << 32;
    return k + (uint64_t)j;
}

__device__ __host__  static uint64_t rng_set_feature_seed(RNG *rng, uint64_t p_190065_, int32_t p_190066_, int32_t p_190067_) {
    uint64_t i = p_190065_ + (uint64_t)p_190066_ + (uint64_t)(10000 * p_190067_);
    //printf("Salt = %" PRIu64 "\n", (uint64_t)p_190066_ + (uint64_t)(10000 * p_190067_));
    rng_set_seed(rng, i);
    return i;
}

__device__ __host__  uint64_t reverse_decoration_seed(uint64_t decorator_seed, int index, int step) {
    return decorator_seed - (uint64_t)index - 10000L * (uint64_t)step;
}

__device__ __host__  static uint64_t rng_set_decoration_seed(RNG *rng, uint64_t world_seed, int32_t x, int32_t z) {
    rng_set_seed(rng, world_seed);

    uint64_t a = rng_next_long(rng) | 1L;
    uint64_t b = rng_next_long(rng) | 1L;

    // printf("the k to recover = %" PRIu64 "\n", (a * (uint64_t)x + b * (uint64_t)z));
    uint64_t k = (a * (uint64_t)x + b * (uint64_t)z) ^ world_seed;
    // printf("real k = %" PRIu64 "\n", k);
    // printf("invert k = %" PRIu64 "\n", k ^ world_seed);
    rng_set_seed(rng, k);
    return k;
}

typedef struct __align__(16) {
    int dx, dz, height;
    bool is_valid;
} Offset;

__device__ __host__  Offset offset_new(int dx, int dz, int height) {
    return (Offset){.dx=dx, .dz=dz, .height=height, .is_valid=true};
}

__device__ __host__  Offset offset_invalid_new() {
    return (Offset){.dx=-1, .dz=-1, .height=-1, .is_valid=false};
}

__device__ __host__  Offset get_position_standard(RNG *rng) {
    int dx = rng_next_int(rng, 16); // spread
    int dz = rng_next_int(rng, 16);

    int i = -144;
    int j = 16;
    int plateau = 0;

    int l = ((j-i) - plateau) / 2;
    int i1 = (j-i) - l;
    int height = i + rng_next_between_inclusive(rng, 0, i1) + rng_next_between_inclusive(rng, 0, l);

    return offset_new(dx, dz, height);
}

__device__ __host__  Offset get_small_diamond_position(RNG *rng, uint64_t chunk_seed) {
    // uint64_t feature_seed = rng_set_feature_seed(rng, chunk_seed, 18, 6);
    // (void)feature_seed;
    
    return get_position_standard(rng);
}

__device__ __host__  Offset get_medium_diamond_position(RNG *rng, uint64_t chunk_seed) {
    // uint64_t feature_seed = rng_set_feature_seed(rng, chunk_seed, 19, 6);
    // (void)feature_seed;
    
    int dx = rng_next_int(rng, 16);
    int dz = rng_next_int(rng, 16);

    int i = -64;
    int j = -4;

    int height = rng_next_between_inclusive(rng, i, j);

    return offset_new(dx, dz, height);
}

__device__ __host__ Offset get_large_diamond_position(RNG *rng, uint64_t chunk_seed) {
    uint64_t feature_seed = rng_set_feature_seed(rng, chunk_seed, 20, 6);
    (void)feature_seed;

    if (!(rng_next_float(rng) < 1.0F / (float)9.0)) {
        return offset_invalid_new();
    }

    return get_position_standard(rng);
}

__device__ __host__  Offset get_buried_diamond_position(RNG *rng, uint64_t chunk_seed) {
    // uint64_t feature_seed = rng_set_feature_seed(rng, chunk_seed, 21, 6);
    // (void)feature_seed;

    return get_position_standard(rng);
}



__device__ __host__  float offset_distance_squared(const Offset *a, const Offset *b) {
    int x1 = a->dx;
    int y1 = a->height;
    int z1 = a->dz;

    int x2 = b->dx;
    int y2 = b->height;
    int z2 = b->dz;

    return ((x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1));
}

#define PI 3.14159265358979

__device__ __host__  bool in_range(int y) {
    return (y > -55) && (y < -6);
}

__device__ __host__  bool get_small_diamond_offsets(RNG *rng, uint64_t chunk_seed, Offset *offsets, size_t *sz) {
    uint64_t feature_seed = rng_set_feature_seed(rng, chunk_seed, 18, 6);
    Offset o = get_small_diamond_position(rng, chunk_seed);
    offsets[*sz] = o;
    (*sz)++;
    return in_range(o.height);
}

__device__ __host__  bool get_medium_diamond_offsets(RNG *rng, uint64_t chunk_seed, Offset *offsets, size_t *sz) {
    uint64_t feature_seed = rng_set_feature_seed(rng, chunk_seed, 19, 6);
    Offset o = get_medium_diamond_position(rng, chunk_seed);
    offsets[*sz] = o;
    (*sz)++;
    return in_range(o.height);
}

__device__ __host__  bool get_buried_diamond_offsets(RNG *rng, uint64_t chunk_seed, Offset *offsets, size_t *sz) {
    uint64_t feature_seed = rng_set_feature_seed(rng, chunk_seed, 21, 6);
    #pragma unroll
    for (int i = 0; i < 4; i++) {
        Offset o = get_buried_diamond_position(rng, chunk_seed);
        
        if (!in_range(o.height)) {
            return false;
        }

        // rng->num_calls = 0;
        offsets[*sz] = o;
        (*sz)++;
        // advance_rng(&rng, o.dx, o.height, o.dz, buried_diamond_size, buried_diamond_discard); 
        // rng->num_calls = 0;
        rng_next_float(rng);
        rng_next_int(rng, 3);
        rng_next_int(rng, 3);

        #pragma unroll
        for (int i = 0; i < 8; i++) {
            rng_next_double(rng);
        }
    }
    return true;
}

__global__ void kernel(uint64_t s, uint64_t *out) {
    uint64_t chunk_seed = blockDim.x * blockIdx.x + threadIdx.x + s;

    RNG rng = rng_new();
    Offset large = get_large_diamond_position(&rng, chunk_seed);
    
    if (!large.is_valid) {
        return;
    }

    Offset offsets[15] = {0};
    size_t sz = 1;
    offsets[0] = large;

    if (!get_small_diamond_offsets(&rng, chunk_seed, offsets, &sz)) {
        return;
    }
    if (!get_medium_diamond_offsets(&rng, chunk_seed, offsets, &sz)) {
        return;
    }
    if (!get_buried_diamond_offsets(&rng, chunk_seed, offsets, &sz)) {
        return;
    }

    const Offset *cmp = &offsets[0];
    #pragma unroll
    for (int i = 1; i < sz; i++) {
        if (offset_distance_squared(cmp, (const Offset *)&offsets[i]) > 9.0) {
            return;
        }
    }
    out[blockDim.x * blockIdx.x + threadIdx.x] = chunk_seed;
}

#include <time.h>
#include <chrono>
using namespace std::chrono;

#ifdef __GNUC__

#include <unistd.h>
#include <sys/time.h>

#endif

/*
    You can add anything you want to checkpoint_vars.
    Be sure to update the checkpointing sections below to reflect the new item in the struct (to save the data into the struct and then to disk)
*/
struct checkpoint_vars {
    unsigned long long offset;
    uint64_t elapsed_chkpoint;
};

uint64_t elapsed_chkpoint = 0;

int main(int argc, char **argv) {

    /*
        The way this has been written, each loop, it calls 32768 * 32 (1048576) kernel threads that each individually run a single seed.
        We refer to these loops as "blocks" of seeds in this code.
        --start defines the starting block (--start 0 begins at seed 0, --start 1 begins at seed 1048576, --start 2 begins at 2097152)
        --end defines the ending block (--end 0 finishes at seed 0, --end 1 finishes at seed 1048576, --end 3 begins at seed 2097152)
        --device defines which GPU ID runs the cuda kernels. You can check this using nvidia-smi if you're running standalone. Otherwise, if you're running on BOINC, this parameter is unneeded on modern clients. Keep it implemented for old clients.
    */
    uint64_t block_min = 0;
    uint64_t block_max = 0;
    uint64_t checked = 0;
    int device = 0;
    for (int i = 1; i < argc; i += 2) {
		const char *param = argv[i];
		if (strcmp(param, "-d") == 0 || strcmp(param, "--device") == 0) {
			device = atoi(argv[i + 1]);
		} else if (strcmp(param, "-s") == 0 || strcmp(param, "--start") == 0) {
			sscanf(argv[i + 1], "%llu", &block_min);
		} else if (strcmp(param, "-e") == 0 || strcmp(param, "--end") == 0) {
			sscanf(argv[i + 1], "%llu", &block_max);
		} 
        else {
			fprintf(stderr,"Unknown parameter: %s\n", param);
        }
    }
    uint64_t offsetStart = 0;
    uint64_t *out;
    //GPU Params
	int blocks = 32768;
	int threads = 32;
    //BOINC
  	#ifdef BOINC

        BOINC_OPTIONS options;
        boinc_options_defaults(options);
	    options.normal_thread_priority = true;
        boinc_init_options(&options);
        APP_INIT_DATA aid;
	    boinc_get_init_data(aid);
        if (aid.gpu_device_num >= 0) {
            //If BOINC client provided us a device ID
		    device = aid.gpu_device_num;
		    fprintf(stderr,"boinc gpu %i gpuindex: %i \n", aid.gpu_device_num, device);
		} else {
            //If BOINC client did not provide us a device ID
            device = -5;
            for (int i = 1; i < argc; i += 2) {
                //Check for a --device flag, just in case we missed it earlier, use it if it's available. For older clients primarily.
              	if(strcmp(argv[i], "--device") == 0){
                    sscanf(argv[i + 1], "%i", &device);
                }
  
            }
            if(device == -5){
                //Something has gone wrong. It pulled from BOINC, got -1. No --device parameter present.
                fprintf(stderr, "Error: No --device parameter provided! Defaulting to device 0...\n");
                device = 0;
            }
		    fprintf(stderr,"stndalone gpuindex %i (aid value: %i)\n", device, aid.gpu_device_num);
	    }   

        FILE *checkpoint_data = boinc_fopen("checkpoint.txt", "rb");
        if(!checkpoint_data){
            //No checkpoint file was found. Proceed from the beginning.
            fprintf(stderr, "No checkpoint to load\n");

        }
        else{
            //Load from checkpoint. You can put any data in data_store that you need to keep between runs of this program.
            boinc_begin_critical_section();
            struct checkpoint_vars data_store;
            fread(&data_store, sizeof(data_store), 1, checkpoint_data);
            offsetStart = data_store.offset;
            elapsed_chkpoint = data_store.elapsed_chkpoint;
            fprintf(stderr, "Checkpoint loaded, task time %d s, seed pos: %llu\n", elapsed_chkpoint, offsetStart);
            fclose(checkpoint_data);
            boinc_end_critical_section();
        }
    #endif
    hipSetDevice(device);
    hipMallocManaged(&out, (blocks * threads) * sizeof(*out));
    for(int i = 0; i < (blocks * threads); i++){
        out[i] = 0;
    }
    auto start = high_resolution_clock::now();
	printf("starting...\n");
    uint64_t checkpointTemp = 0;
    FILE* seedsout = fopen("seeds.txt", "w+");
    for (uint64_t s = (uint64_t)block_min + offsetStart; s < (uint64_t)block_max; s++) {
        //Call GPU kernel
        kernel<<<blocks, threads>>>(blocks * threads * s, out);
        GPU_ASSERT(hipPeekAtLastError());
        GPU_ASSERT(hipDeviceSynchronize());  
        //Check error from GPU driver, if any
        checkpointTemp += 1;
        #ifdef BOINC
        if(checkpointTemp >= 15 || boinc_time_to_checkpoint()){
            //Checkpointing for BOINC
            auto checkpoint_end = high_resolution_clock::now();
            auto duration = duration_cast<milliseconds>(checkpoint_end - start);
            boinc_begin_critical_section(); // Boinc should not interrupt this
            
            // Checkpointing section below
            boinc_delete_file("checkpoint.txt"); // Don't touch, same func as normal fdel
            FILE *checkpoint_data = boinc_fopen("checkpoint.txt", "wb");
            struct checkpoint_vars data_store;
            data_store.offset = s - block_min;
            data_store.elapsed_chkpoint = elapsed_chkpoint + duration.count();
            fwrite(&data_store, sizeof(data_store), 1, checkpoint_data);
            fclose(checkpoint_data);
            checkpointTemp = 0;
            boinc_end_critical_section();
            boinc_checkpoint_completed(); // Checkpointing completed
        }
        //Update boinc client with percentage
        double frac = (double)(s+1 - block_min) / (double)(block_max - block_min);
        boinc_fraction_done(frac);

        #endif
        for (unsigned long long i = 0; i < blocks * threads; i++){
            if(out[i] > 0){
			    fprintf(seedsout,"%llu\n", out[i]);
                out[i] = 0;
                //Grab values from `out` buffer and print to seedsout
                //Set to 0 after to reset
            }

		}
		fflush(seedsout);

    }


    /*
        The end. This prints speed information to stderr.txt - which will be uploaded to the BOINC server, or it can be reviewed locally in a standsalone run.
    */
    auto end = high_resolution_clock::now();
    auto duration = duration_cast<milliseconds>(end - start);
    checked = blocks*threads*(block_max - block_min);
    fprintf(stderr, "checked = %" PRIu64 "\n", checked);
    fprintf(stderr, "time taken = %f\n", (double)duration.count()/1000.0);

	double seeds_per_second = checked / ((double)duration.count()/1000.0);
	double speedup = seeds_per_second / 199000;
	fprintf(stderr, "seeds per second: %f\n", seeds_per_second);
	fprintf(stderr, "speedup: %fx\n", speedup);
    boinc_finish(0);
}
